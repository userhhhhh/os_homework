#include "hip/hip_runtime.h"
// nccl_multi_node.cu
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <nccl.h>
#include <mpi.h>
#include <sys/time.h>

// 错误检查宏
#define CUDA_CHECK(cmd) do {                         \
  hipError_t err = cmd;                             \
  if (err != hipSuccess) {                          \
    printf("CUDA error %s:%d: '%s'\n",               \
        __FILE__, __LINE__, hipGetErrorString(err));\
    exit(EXIT_FAILURE);                              \
  }                                                  \
} while(0)

#define NCCL_CHECK(cmd) do {                         \
  ncclResult_t res = cmd;                            \
  if (res != ncclSuccess) {                          \
    printf("NCCL error %s:%d: '%s'\n",               \
        __FILE__, __LINE__, ncclGetErrorString(res));\
    exit(EXIT_FAILURE);                              \
  }                                                  \
} while(0)

#define MPI_CHECK(cmd) do {                          \
  int err = cmd;                                     \
  if (err != MPI_SUCCESS) {                          \
    char errstr[MPI_MAX_ERROR_STRING];               \
    int len;                                         \
    MPI_Error_string(err, errstr, &len);             \
    printf("MPI error %s:%d: '%s'\n",                \
           __FILE__, __LINE__, errstr);              \
    MPI_Abort(MPI_COMM_WORLD, 1);                    \
    exit(EXIT_FAILURE);                              \
  }                                                  \
} while(0)

// 获取当前时间（微秒）
double get_time_us() {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec * 1000000 + tv.tv_usec;
}

int main(int argc, char* argv[]) {
    // 初始化MPI
    MPI_CHECK(MPI_Init(&argc, &argv));
    
    // 获取MPI进程信息
    int rank, world_size;
    MPI_CHECK(MPI_Comm_rank(MPI_COMM_WORLD, &rank));
    MPI_CHECK(MPI_Comm_size(MPI_COMM_WORLD, &world_size));
    
    // 打印节点信息
    char hostname[1024];
    gethostname(hostname, 1024);
    printf("Rank %d 运行在 %s\n", rank, hostname);
    
    // 获取可用的GPU数量
    int deviceCount;
    CUDA_CHECK(hipGetDeviceCount(&deviceCount));
    
    // 确保进程数不超过GPU数量
    if (world_size > deviceCount) {
        if (rank == 0) {
            printf("错误：进程数(%d)大于GPU数量(%d)\n", world_size, deviceCount);
        }
        MPI_Abort(MPI_COMM_WORLD, 1);
        exit(EXIT_FAILURE);
    }
    
    // 每个进程使用不同的GPU
    CUDA_CHECK(hipSetDevice(rank % deviceCount));
    printf("Rank %d 使用 GPU %d\n", rank, rank % deviceCount);
    
    // 创建NCCL唯一ID并广播给所有进程
    ncclUniqueId nccl_id;
    if (rank == 0) {
        NCCL_CHECK(ncclGetUniqueId(&nccl_id));
    }
    MPI_CHECK(MPI_Bcast(&nccl_id, sizeof(nccl_id), MPI_BYTE, 0, MPI_COMM_WORLD));
    
    // 创建NCCL通信器
    ncclComm_t comm;
    NCCL_CHECK(ncclCommInitRank(&comm, world_size, nccl_id, rank));
    
    printf("Rank %d: NCCL通信器初始化成功\n", rank);
    
    // 同步所有进程
    MPI_CHECK(MPI_Barrier(MPI_COMM_WORLD));
    
    // 数据大小（从1MB到256MB）
    size_t sizes[] = {
        1 << 20,      // 1 MB
        4 << 20,      // 4 MB
        16 << 20,     // 16 MB
        64 << 20,     // 64 MB
        256 << 20     // 256 MB
    };
    
    for (int s = 0; s < sizeof(sizes)/sizeof(sizes[0]); s++) {
        size_t size = sizes[s];
        size_t count = size / sizeof(float);
        
        if (rank == 0) {
            printf("\n测试数据大小: %.2f MB\n", size / (1024.0 * 1024.0));
        }
        
        // 分配GPU内存
        float *sendbuf, *recvbuf;
        CUDA_CHECK(hipMalloc(&sendbuf, size));
        CUDA_CHECK(hipMalloc(&recvbuf, size));
        
        // 分配主机内存用于验证
        float *h_sendbuf = (float*)malloc(size);
        float *h_recvbuf = (float*)malloc(size);
        
        // 初始化数据
        if (rank == 0) {
            for (size_t i = 0; i < count; i++) {
                h_sendbuf[i] = (float)rand() / RAND_MAX;
            }
        } else {
            memset(h_sendbuf, 0, size);
        }
        
        CUDA_CHECK(hipMemcpy(sendbuf, h_sendbuf, size, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemset(recvbuf, 0, size));
        
        // 创建CUDA流
        hipStream_t stream;
        CUDA_CHECK(hipStreamCreate(&stream));
        
        // 同步所有进程
        MPI_CHECK(MPI_Barrier(MPI_COMM_WORLD));
        
        // -------------------------------------
        // 测试广播操作 (Broadcast)
        // -------------------------------------
        if (rank == 0) {
            printf("测试广播操作 (Broadcast):\n");
        }
        
        // 热身
        for (int i = 0; i < 3; i++) {
            NCCL_CHECK(ncclBroadcast(sendbuf, sendbuf, count, ncclFloat, 0, comm, stream));
            CUDA_CHECK(hipStreamSynchronize(stream));
        }
        
        // 同步所有进程
        MPI_CHECK(MPI_Barrier(MPI_COMM_WORLD));
        
        // 计时开始
        double start_time = get_time_us();
        
        // 执行NCCL广播操作
        NCCL_CHECK(ncclBroadcast(sendbuf, sendbuf, count, ncclFloat, 0, comm, stream));
        CUDA_CHECK(hipStreamSynchronize(stream));
        
        // 计时结束
        double end_time = get_time_us();
        double elapsed = end_time - start_time;
        
        // 收集所有进程的时间
        double max_time;
        MPI_CHECK(MPI_Reduce(&elapsed, &max_time, 1, MPI_DOUBLE, MPI_MAX, 0, MPI_COMM_WORLD));
        
        // 将结果拷贝回主机内存以验证
        CUDA_CHECK(hipMemcpy(h_recvbuf, sendbuf, size, hipMemcpyDeviceToHost));
        
        // 仅在rank 0上输出性能结果
        if (rank == 0) {
            // 计算带宽
            double bandwidth = (size * (world_size-1)) / (max_time / 1000000.0) / (1024.0 * 1024.0 * 1024.0);
            printf("NCCL广播性能：时间 = %.3f ms, 吞吐量 = %.3f GB/s\n", 
                   max_time / 1000.0, bandwidth);
        }
        
        // 同步所有进程
        MPI_CHECK(MPI_Barrier(MPI_COMM_WORLD));
        
        // -------------------------------------
        // 测试规约操作 (AllReduce)
        // -------------------------------------
        if (rank == 0) {
            printf("\n测试规约操作 (AllReduce):\n");
        }
        
        // 重新初始化数据
        for (size_t i = 0; i < count; i++) {
            h_sendbuf[i] = (float)(rank + 1) / world_size;
        }
        
        CUDA_CHECK(hipMemcpy(sendbuf, h_sendbuf, size, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemset(recvbuf, 0, size));
        
        // 热身
        for (int i = 0; i < 3; i++) {
            NCCL_CHECK(ncclAllReduce(sendbuf, recvbuf, count, ncclFloat, ncclSum, comm, stream));
            CUDA_CHECK(hipStreamSynchronize(stream));
        }
        
        // 同步所有进程
        MPI_CHECK(MPI_Barrier(MPI_COMM_WORLD));
        
        // 计时开始
        start_time = get_time_us();
        
        // 执行NCCL AllReduce操作
        NCCL_CHECK(ncclAllReduce(sendbuf, recvbuf, count, ncclFloat, ncclSum, comm, stream));
        CUDA_CHECK(hipStreamSynchronize(stream));
        
        // 计时结束
        end_time = get_time_us();
        elapsed = end_time - start_time;
        
        // 收集所有进程的时间
        MPI_CHECK(MPI_Reduce(&elapsed, &max_time, 1, MPI_DOUBLE, MPI_MAX, 0, MPI_COMM_WORLD));
        
        // 将结果拷贝回主机内存以验证
        CUDA_CHECK(hipMemcpy(h_recvbuf, recvbuf, size, hipMemcpyDeviceToHost));
        
        // 验证AllReduce结果是否正确
        float expected_sum = 0;
        for (int i = 0; i < world_size; i++) {
            expected_sum += (float)(i + 1) / world_size;
        }
        
        bool allreduce_correct = true;
        for (size_t i = 0; i < count; i++) {
            if (fabsf(h_recvbuf[i] - expected_sum) > 1e-5) {
                printf("Rank %d: AllReduce验证失败：位置 %zu 值为 %f, 期望值 %f\n", 
                       rank, i, h_recvbuf[i], expected_sum);
                allreduce_correct = false;
                break;
            }
        }
        
        if (allreduce_correct && rank == 0) {
            printf("AllReduce验证成功：所有GPU的结果等于预期的总和\n");
        }
        
        // 仅在rank 0上输出性能结果
        if (rank == 0) {
            // 计算带宽 (2x因为数据双向流动)
            double bandwidth = (2.0 * size * world_size) / (max_time / 1000000.0) / (1024.0 * 1024.0 * 1024.0);
            printf("NCCL AllReduce性能：时间 = %.3f ms, 吞吐量 = %.3f GB/s\n", 
                   max_time / 1000.0, bandwidth);
        }
        
        // 释放资源
        CUDA_CHECK(hipFree(sendbuf));
        CUDA_CHECK(hipFree(recvbuf));
        free(h_sendbuf);
        free(h_recvbuf);
        CUDA_CHECK(hipStreamDestroy(stream));
    }
    
    // 销毁NCCL通信器
    ncclCommDestroy(comm);
    
    // 结束MPI
    MPI_Finalize();
    
    return 0;
}