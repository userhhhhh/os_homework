// nccl_test.cu
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <nccl.h>
#include <unistd.h>
#include <time.h>
#include <sys/time.h>
#include <pthread.h>
#include <math.h>

// 错误检查宏
#define CUDA_CHECK(cmd) do {                         \
  hipError_t err = cmd;                             \
  if (err != hipSuccess) {                          \
    printf("CUDA error %s:%d: '%s'\n",               \
        __FILE__, __LINE__, hipGetErrorString(err));\
    exit(EXIT_FAILURE);                              \
  }                                                  \
} while(0)

#define NCCL_CHECK(cmd) do {                         \
  ncclResult_t res = cmd;                            \
  if (res != ncclSuccess) {                          \
    printf("NCCL error %s:%d: '%s'\n",               \
        __FILE__, __LINE__, ncclGetErrorString(res));\
    exit(EXIT_FAILURE);                              \
  }                                                  \
} while(0)

// 线程参数结构体
typedef struct {
    int dev;
    float* sendbuff;
    float* recvbuff;
    size_t count;
    ncclComm_t comm;
    int root;
    int op_type; // 0: broadcast, 1: allreduce
} thread_args_t;

// 广播函数实现
ncclResult_t nccl_broadcast_data(void* data, size_t count, int root, ncclComm_t comm) {
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));
    ncclResult_t result = ncclBroadcast(data, data, count, ncclFloat, root, comm, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipStreamDestroy(stream));
    return result;
}

// AllReduce函数实现
ncclResult_t nccl_allreduce_data(void* sendbuff, void* recvbuff, size_t count, ncclComm_t comm) {
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));
    ncclResult_t result = ncclAllReduce(sendbuff, recvbuff, count, ncclFloat, ncclSum, comm, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipStreamDestroy(stream));
    return result;
}

// 线程执行函数
void* nccl_thread_func(void* arg) {
    thread_args_t* args = (thread_args_t*)arg;
    CUDA_CHECK(hipSetDevice(args->dev));
    if (args->op_type == 0) {
        nccl_broadcast_data(args->sendbuff, args->count, args->root, args->comm);
    } else {
        nccl_allreduce_data(args->sendbuff, args->recvbuff, args->count, args->comm);
    }
    return NULL;
}

// 获取当前时间（微秒）
double get_time_us() {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec * 1000000 + tv.tv_usec;
}

int main(int argc, char* argv[]) {
    // 设置随机种子
    srand(time(NULL));
    
    // 获取GPU数量
    int nDev = 0;
    CUDA_CHECK(hipGetDeviceCount(&nDev));
    printf("Found %d GPUs\n", nDev);
    
    // 如果不足两个GPU，则退出
    if (nDev < 2) {
        printf("需要至少2个GPU才能进行测试\n");
        return 0;
    }
    
    // 设置使用两个GPU
    nDev = 2;  // 只使用2个GPU进行测试
    
    // 为每个GPU分配ID
    int devs[2] = {0, 1};
    
    // 创建NCCL通信器
    ncclComm_t comms[2];
    NCCL_CHECK(ncclCommInitAll(comms, nDev, devs));
    printf("NCCL通信器初始化成功\n");
    
    // 数据大小（从1MB到256MB）
    size_t sizes[] = {
        1 << 20,      // 1 MB
        4 << 20,      // 4 MB
        16 << 20,     // 16 MB
        64 << 20,     // 64 MB
        256 << 20     // 256 MB
    };
    
    for (int s = 0; s < sizeof(sizes)/sizeof(sizes[0]); s++) {
        size_t size = sizes[s];
        size_t count = size / sizeof(float);
        
        printf("\n测试数据大小: %.2f MB\n", size / (1024.0 * 1024.0));
        
        // 为每个GPU分配内存和准备数据
        float** sendbuff = (float**)malloc(nDev * sizeof(float*));
        float** recvbuff = (float**)malloc(nDev * sizeof(float*));
        float** hostbuff = (float**)malloc(nDev * sizeof(float*));
        
        for (int i = 0; i < nDev; i++) {
            CUDA_CHECK(hipSetDevice(i));
            CUDA_CHECK(hipMalloc(&sendbuff[i], size));
            CUDA_CHECK(hipMalloc(&recvbuff[i], size));
            hostbuff[i] = (float*)malloc(size);
            
            // 初始化数据：对于设备0填充随机数，其他设备填充0
            if (i == 0) {
                for (size_t j = 0; j < count; j++) {
                    hostbuff[i][j] = (float)rand() / RAND_MAX;
                }
            } else {
                memset(hostbuff[i], 0, size);
            }
            
            // 将主机数据拷贝到GPU
            CUDA_CHECK(hipMemcpy(sendbuff[i], hostbuff[i], size, hipMemcpyHostToDevice));
            CUDA_CHECK(hipMemset(recvbuff[i], 0, size));
        }
        
        // -------------------------------------
        // 测试广播操作 (Broadcast)
        // -------------------------------------
        printf("测试广播操作 (Broadcast):\n");
        
        // 执行广播操作前同步所有设备
        for (int i = 0; i < nDev; i++) {
            CUDA_CHECK(hipSetDevice(i));
            CUDA_CHECK(hipDeviceSynchronize());
        }
        
        // 计时开始
        double start_time = get_time_us();
        
        // 执行NCCL广播操作 (根节点为0) - 使用多线程
        pthread_t threads[nDev];
        thread_args_t targs[nDev];
        
        for (int i = 0; i < nDev; i++) {
            targs[i].dev = i;
            targs[i].sendbuff = sendbuff[i];
            targs[i].recvbuff = NULL; // 广播不需要recvbuff
            targs[i].count = count;
            targs[i].comm = comms[i];
            targs[i].root = 0;
            targs[i].op_type = 0; // 0: broadcast
            pthread_create(&threads[i], NULL, nccl_thread_func, &targs[i]);
        }
        
        // 等待所有线程完成
        for (int i = 0; i < nDev; i++) {
            pthread_join(threads[i], NULL);
        }
        
        // 确保所有操作完成
        for (int i = 0; i < nDev; i++) {
            CUDA_CHECK(hipSetDevice(i));
            CUDA_CHECK(hipDeviceSynchronize());
        }
        
        // 计时结束
        double end_time = get_time_us();
        double elapsed = end_time - start_time;
        
        // 将结果拷贝回主机内存以验证
        for (int i = 0; i < nDev; i++) {
            CUDA_CHECK(hipSetDevice(i));
            CUDA_CHECK(hipMemcpy(hostbuff[i], sendbuff[i], size, hipMemcpyDeviceToHost));
        }
        
        // 验证广播结果是否一致
        bool broadcast_correct = true;
        for (int i = 1; i < nDev; i++) {
            for (size_t j = 0; j < count; j++) {
                if (hostbuff[0][j] != hostbuff[i][j]) {
                    printf("广播验证失败：GPU %d 与 GPU 0 的数据不一致\n", i);
                    broadcast_correct = false;
                    break;
                }
            }
            if (!broadcast_correct) break;
        }
        
        if (broadcast_correct) {
            printf("广播验证成功：所有GPU的数据一致\n");
        }
        
        // 计算带宽和延迟
        double bandwidth = (size * (nDev-1)) / (elapsed / 1000000.0) / (1024.0 * 1024.0 * 1024.0);
        printf("广播性能：时间 = %.3f ms, 吞吐量 = %.3f GB/s\n", 
               elapsed / 1000.0, bandwidth);
        
        // -------------------------------------
        // 测试规约操作 (AllReduce)
        // -------------------------------------
        printf("\n测试规约操作 (AllReduce):\n");
        
        // 重新准备数据
        for (int i = 0; i < nDev; i++) {
            CUDA_CHECK(hipSetDevice(i));
            
            // 为每个GPU初始化不同的数据
            for (size_t j = 0; j < count; j++) {
                hostbuff[i][j] = (float)(i + 1) / nDev; // 简单的不同值
            }
            
            CUDA_CHECK(hipMemcpy(sendbuff[i], hostbuff[i], size, hipMemcpyHostToDevice));
        }
        
        // 执行AllReduce操作前同步所有设备
        for (int i = 0; i < nDev; i++) {
            CUDA_CHECK(hipSetDevice(i));
            CUDA_CHECK(hipDeviceSynchronize());
        }
        
        // 计时开始
        start_time = get_time_us();
        
        // 执行NCCL AllReduce操作（使用多线程）
        for (int i = 0; i < nDev; i++) {
            targs[i].dev = i;
            targs[i].sendbuff = sendbuff[i];
            targs[i].recvbuff = recvbuff[i];
            targs[i].count = count;
            targs[i].comm = comms[i];
            targs[i].op_type = 1; // 1: allreduce
            pthread_create(&threads[i], NULL, nccl_thread_func, &targs[i]);
        }
        
        // 等待所有线程完成
        for (int i = 0; i < nDev; i++) {
            pthread_join(threads[i], NULL);
        }
        
        // 确保所有操作完成
        for (int i = 0; i < nDev; i++) {
            CUDA_CHECK(hipSetDevice(i));
            CUDA_CHECK(hipDeviceSynchronize());
        }
        
        // 计时结束
        end_time = get_time_us();
        elapsed = end_time - start_time;
        
        // 将结果拷贝回主机内存以验证
        for (int i = 0; i < nDev; i++) {
            CUDA_CHECK(hipSetDevice(i));
            CUDA_CHECK(hipMemcpy(hostbuff[i], recvbuff[i], size, hipMemcpyDeviceToHost));
        }
        
        // 验证AllReduce结果是否正确 - 所有值应该是 (1+2)/2 = 1.5
        bool allreduce_correct = true;
        float expected_sum = 0;
        for (int i = 0; i < nDev; i++) {
            expected_sum += (float)(i + 1) / nDev;
        }
        
        for (int i = 0; i < nDev; i++) {
            for (size_t j = 0; j < count; j++) {
                if (fabs(hostbuff[i][j] - expected_sum) > 1e-5) {
                    printf("AllReduce验证失败：GPU %d 的结果与预期不符\n", i);
                    allreduce_correct = false;
                    break;
                }
            }
            if (!allreduce_correct) break;
        }
        
        if (allreduce_correct) {
            printf("AllReduce验证成功：所有GPU的结果等于预期的总和\n");
        }
        
        // 计算带宽和延迟
        bandwidth = (2.0 * size * nDev) / (elapsed / 1000000.0) / (1024.0 * 1024.0 * 1024.0);
        printf("AllReduce性能：时间 = %.3f ms, 吞吐量 = %.3f GB/s\n", 
               elapsed / 1000.0, bandwidth);
               
        // 释放资源
        for (int i = 0; i < nDev; i++) {
            CUDA_CHECK(hipSetDevice(i));
            CUDA_CHECK(hipFree(sendbuff[i]));
            CUDA_CHECK(hipFree(recvbuff[i]));
            free(hostbuff[i]);
        }
        free(sendbuff);
        free(recvbuff);
        free(hostbuff);
    }
    
    // 销毁NCCL通信器
    for (int i = 0; i < nDev; i++) {
        ncclCommDestroy(comms[i]);
    }
    
    return 0;
}